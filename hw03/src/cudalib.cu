#include "hip/hip_runtime.h"
#include "utils.cuh"
#include "const.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ void get_xij_d(int i, int j, int dim, double *xarr, double *xij, int N){
    for (int k = 0; k < dim; k++){
        xij[k] = xarr[j*dim+k] - xarr[i*dim+k];
    }
}

__device__ void partition_d(int nsteps, int size, int idx, int *start_ptr, int *end_ptr){
    *start_ptr = nsteps / size * idx;
    *end_ptr = nsteps / size * (idx+1);
    if (idx+1==size) *end_ptr = nsteps;
}

__device__ double norm_d(double *x, int dim){
    double r = 0;
    for (int i = 0; i < dim; i++){
        r += x[i]*x[i];
    }
    r = sqrt(r);
    return r;
}

__device__ void vec_add_d(double *a, double *b, double *c, 
    double fac1, double fac2, int dim){
    for (int i = 0; i < dim; i++){
        a[i] = fac1*b[i] + fac2*c[i];
    }
}


__global__ void vec_add_cu(double *a, double *b, double *c, int dim){
    int size = blockDim.x * gridDim.x;
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    int start_idx, end_idx;
    partition_d(dim, size, idx, &start_idx, &end_idx);
    for (int i = start_idx; i < end_idx; i++){
        a[i] = b[i] + c[i];
    }
}

__global__ void vec_sub_cu(double *a, double *b, double *c, int dim){
    int size = blockDim.x * gridDim.x;
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    int start_idx, end_idx;
    partition_d(dim, size, idx, &start_idx, &end_idx);
    for (int i = start_idx; i < end_idx; i++){
        a[i] = b[i] - c[i];
    }
}

__global__ void gather_dx_cu(double *a, double *b, double *c, int dim){
    int size = blockDim.x * gridDim.x;
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    int start_idx, end_idx;
    partition_d(dim, size, idx, &start_idx, &end_idx);
    for (int i = start_idx; i < end_idx; i++){
        a[i] += b[i] - c[i];
    }
}

__global__ void verlet_at2_cu(const int dim, double *marr, double *xarr, double *xarr0,
    double *dxarr, double dt, double G, int N, double cut){
    int size = blockDim.x * gridDim.x;
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    int start_idx, end_idx;
    partition_d(N, size, idx, &start_idx, &end_idx);
    // printf("%d %d\n", start_idx, end_idx);
    // TODO: check later
    for (int i = start_idx; i < end_idx; i++){
        double tmp0 = 0.0;
        double tmp1 = 0.0;
        for (int j = 0; j < N; j++){
            if (j!=i){
            // get xij
            double xij0 = xarr[j*dim+0] - xarr[i*dim+0];
            double xij1 = xarr[j*dim+1] - xarr[i*dim+1];
            // compute rij
            double rij = sqrt(xij0*xij0 + xij1*xij1);
            double fac = 1.0;
            if (rij < cut) {
                rij = cut;
            }
            tmp0 += xij0 * G/(rij*rij*rij) * marr[j]*dt*dt;
            tmp1 += xij1 * G/(rij*rij*rij) * marr[j]*dt*dt;
            }
        }
        dxarr[i*dim + 0] = tmp0;
        dxarr[i*dim + 1] = tmp1;
    }
}

__global__ void print_arr_cu(double *arr, int dim){
    printf("parr_cu1\n");
    for (int i = 0; i < dim; i++){
        printf("%f ", arr[i]);
    }
    printf("\n");
    printf("parr_cu2\n");
}

// cuda initialize program
void initialize_cu(double *marr, double *xarr, int N, int dim, int Tx, int Ty){
    printf("cuda initialize\n");
    // cuda parameters
    Tx_cu = Tx;
    Ty_cu = Ty;
    // cuda memory allocation
    gpuErrchk( hipMalloc((void **) &marr_d, sizeof(double)*N));
    gpuErrchk( hipMalloc((void **) &xarr_d, sizeof(double)*N*dim));
    gpuErrchk( hipMalloc((void **) &xarr0_d, sizeof(double)*N*dim));
    gpuErrchk( hipMalloc((void **) &dxarr_d, sizeof(double)*N*dim));
    // copy
    gpuErrchk( hipMemcpy(marr_d, marr, sizeof(double)*N, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(xarr_d, xarr, sizeof(double)*N*dim, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(xarr0_d, xarr, sizeof(double)*N*dim, hipMemcpyHostToDevice) );
    
    // print check: passed
    // print_arr_cu<<<1,1>>>(marr_d, N);    
    hipDeviceSynchronize();
}

// switch pointers
__global__ void swap(double * &a, double * &b){
    double *tmp = a;
    a = b;
    b = tmp;
}

// verlet cuda callee
void compute_cu(double *xarr, int nsteps, int N, int dim, double G, double dt, double cut){
    // verlet cuda main program
    double *tmp;
    hipMemset(dxarr_d, 0x00, sizeof(double)*N*dim);
    verlet_at2_cu<<<Tx_cu,Ty_cu>>>(dim, marr_d, xarr_d, xarr0_d, dxarr_d, dt, G, N, cut); // dx: acc
    gather_dx_cu<<<Tx_cu,Ty_cu>>>(dxarr_d, xarr_d, xarr0_d, N*dim);
    tmp = xarr_d;
    xarr_d = xarr0_d;
    xarr0_d = tmp;
    vec_add_cu<<<Tx_cu,Ty_cu>>>(xarr_d, xarr0_d, dxarr_d, N*dim);

    // hipDeviceSynchronize();
    // hipMemcpy(xarr, xarr_d, sizeof(double)*N*dim, hipMemcpyDeviceToHost);

    #ifdef GUI
    // copy x to host
    hipMemcpy(xarr, xarr_d, sizeof(double)*N*dim, hipMemcpyDeviceToHost);
    #endif
}

// cuda finalize program
void finalize_cu(){
    // free
    printf("cuda finalize\n");
    gpuErrchk( hipFree(marr_d) );
    gpuErrchk( hipFree(xarr_d) );
    gpuErrchk( hipFree(xarr0_d) );
    gpuErrchk( hipFree(dxarr_d) );
}

