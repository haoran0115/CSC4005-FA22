#include "hip/hip_runtime.h"
#include "utils.cuh"
#include "const.cuh"
#define BLOCK_SIZE 1024

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ void partition_d(int nsteps, int size, int idx, int *start_ptr, int *end_ptr){
    *start_ptr = nsteps / size * idx;
    *end_ptr = nsteps / size * (idx+1);
    if (idx+1==size) *end_ptr = nsteps;
}

__global__ void print_arr_cu(float *arr, int dim){
    for (int i = 0; i < dim; i++){
        printf("%f ", arr[i]);
    }
    printf("\n");
}

__device__ void print_arr_d(float *arr, int dim){
    for (int i = 0; i < dim; i++){
        printf("%f ", arr[i]);
    }
    printf("\n");
}

void initialize_cu(float *temp_arr, float *temp_arr0, bool *fire_arr,
    float *x_arr, float *y_arr, int DIM, float T_fire, int Tx, int Ty){
    printf("CUDA initialization\n");
    // cuda parameters
    DIM_d = DIM;
    T_fire_d = T_fire;
    Tx_d = Tx;
    Ty_d = Ty;
    // cuda memory allocation
    gpuErrchk( hipMalloc((void **)&temp_arr_d, sizeof(float)*DIM*DIM) );
    gpuErrchk( hipMalloc((void **)&temp_arr0_d, sizeof(float)*DIM*DIM) );
    gpuErrchk( hipMalloc((void **)&fire_arr_d, sizeof(bool)*DIM*DIM) );
    // cuda memory copy
    gpuErrchk( hipMemcpy(temp_arr_d, temp_arr, sizeof(float)*DIM*DIM, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(temp_arr0_d, temp_arr0, sizeof(float)*DIM*DIM, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(fire_arr_d, fire_arr, sizeof(bool)*DIM*DIM, hipMemcpyHostToDevice) );
    // synchronize
    hipDeviceSynchronize();
}

void finalize_cu(){
    printf("CUDA finalization\n");
    // cuda free
    gpuErrchk( hipFree(temp_arr_d) );
    gpuErrchk( hipFree(temp_arr0_d) );
    gpuErrchk( hipFree(fire_arr_d) );
    // synchronize
    hipDeviceSynchronize();
}

__global__ void update_cu_callee(float *temp_arr, float *temp_arr0, bool *fire_arr,
    float *x_arr, float *y_arr, int DIM, float T_fire){
    int start_idx, end_idx;
    int size = blockDim.x * gridDim.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    partition_d(DIM-2, size, idx, &start_idx, &end_idx);
    for (int i = start_idx+1; i < end_idx+1; i++){
    for (int j = 1; j < DIM-1; j++){
        float xw, xa, xs, xd; // w: up; a: left; s: down; d: right
        xw = temp_arr0[i*DIM+j+1];
        xa = temp_arr0[(i-1)*DIM+j];
        xs = temp_arr0[i*DIM+j-1];
        xd = temp_arr0[(i+1)*DIM+j];
        temp_arr[i*DIM+j] = (xw + xa + xs + xd) / 4;
        if (fire_arr[i*DIM+j])
            temp_arr[i*DIM+j] = T_fire;
    }}
}

__global__ void foo(float *arr, int DIM){
    for (int i = 0; i < DIM; i++)
        arr[i] = 0;
}

void update_cu(float *temp_arr){
    update_cu_callee<<<4,4>>>(temp_arr_d, temp_arr0_d, fire_arr_d, 
        NULL, NULL, DIM_d, T_fire_d);
    hipDeviceSynchronize();

    // copy data to host
    gpuErrchk( hipMemcpy(temp_arr, temp_arr_d, sizeof(float)*DIM_d*DIM_d, hipMemcpyDeviceToHost) );
    hipDeviceSynchronize();

    // switch pointers
    float *tmp = temp_arr_d;
    temp_arr_d = temp_arr0_d;
    temp_arr0_d = tmp;

    // synchronize
    hipDeviceSynchronize();
}


